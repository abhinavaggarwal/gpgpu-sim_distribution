
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__global__ void saxpy(unsigned num_rd_streams, unsigned num_wr_streams, unsigned addr1, unsigned addr2, unsigned addr3, unsigned addr4, unsigned addr5, unsigned addr6, unsigned addr7, unsigned addr8, int dummy)
{
	__shared__ float A[1000];
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	for (int i = 0; i < 1000 - 8; i += 8) {
		A[id + 8*i*dummy] = i + 8;
		A[id + 1*i*dummy] = i + 1;
		A[id + 2*i*dummy] = i + 2;
		A[id + 3*i*dummy] = i + 3;
		A[id + 4*i*dummy] = i + 4;
		A[id + 5*i*dummy] = i + 5;
		A[id + 6*i*dummy] = i + 6;
		A[id + 7*i*dummy] = i + 7;
	}
}

int main(int argc, char *argv[])
{
	int N = 1000;
	float *d_x = (float *)100;
	float *h_x;
	h_x = (float *)malloc(N*8*sizeof(float));
	hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice);
	saxpy<<<1, 8>>>(0, 8, 100, 4100, 8100, 12100, 16100, 20100, 24100, 28100, atoi(argv[1]));
	hipMemcpy(h_x, d_x, N*8*sizeof(float), hipMemcpyDeviceToHost);
	for (unsigned i = 0 ; i < 8000 ; i++) {
		printf("%f\n", *(h_x + i));
	}	
}
