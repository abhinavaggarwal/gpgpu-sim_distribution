
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__global__ void saxpy(unsigned num_streams, unsigned addr1, unsigned addr2, unsigned addr3, unsigned addr4, unsigned addr5, unsigned addr6, unsigned addr7, unsigned addr8, int dummy, float *x)
{
	__shared__ float A[1000];
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	float a = 0, b = 0, c = 0, d = 0, e = 0, f = 0, g = 0, h = 0;
	for (int i = 0; i < 1000 - 8; i += 8) {
		a = A[id + 8*i*dummy];
		b = A[id + 1*i*dummy];
		c = A[id + 2*i*dummy];
		d = A[id + 3*i*dummy];
		e = A[id + 4*i*dummy];
		f = A[id + 5*i*dummy];
		g = A[id + 6*i*dummy];
		h = A[id + 7*i*dummy];
	}
	x[id] = a + b + c + d + e + f + g + h;
}

int main(int argc, char *argv[])
{
	int N = 1000;
	// Perform SAXPY on 1M elements
	float *h_x = (float *)malloc(N*sizeof(float));
	float *d_x = (float *)100;
	float *d_x_copy;
	hipMalloc(&d_x_copy, N*sizeof(float));
	// cudaMalloc(&d_x, 2*sizeof(float));
	for (int i = 1 ; i <= N ; i++)
		h_x[i-1] = (float)i;
	hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice);
	float *h_dummy = (float *)malloc(sizeof(float));
	float *d_dummy;
	hipMalloc(&d_dummy, 8*sizeof(float));
	saxpy<<<1, 8>>>(8, 100, 100, 100, 100, 100, 100, 100, 100, atoi(argv[1]), d_dummy);
	//cudaMemcpy(h_dummy, d_dummy, sizeof(float), cudaMemcpyDeviceToHost);
	printf("%f\n", *h_dummy);
}
