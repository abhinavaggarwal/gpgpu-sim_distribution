
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__global__ void saxpy(int n, float *x)
{
  __shared__ float A[1000];
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  float a = 0, b = 0, c = 0, d = 0, e = 0, f = 0, g = 0, h = 0;
  if (id == 0) {
	for (int i = 0 ; i < 1000 ; i += 8) {
		a = A[i];
		b = A[i + 1];
		c = A[i + 2];
		d = A[i + 3];
		e = A[i + 4];
		f = A[i + 5];
		g = A[i + 6];
		h = A[i + 7];
	}
	*x = a + b + c + d + e + f + g + h;
  }
}

int main(void)
{
  int N = 1000;
  // Perform SAXPY on 1M elements
  float *h_x = (float *)malloc(N*sizeof(float));
  float *d_x = (float *)100;
  float *d_x_copy;
  hipMalloc(&d_x_copy, N*sizeof(float));
  // cudaMalloc(&d_x, 2*sizeof(float));
  for (int i = 1 ; i <= N ; i++)
	  h_x[i-1] = (float)i;
  hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice);
  float *h_dummy = (float *)malloc(sizeof(float));
  float *d_dummy;
  hipMalloc(&d_dummy, sizeof(float));
  saxpy<<<1, 8>>>(N, d_dummy);
  hipMemcpy(h_dummy, d_dummy, sizeof(float), hipMemcpyDeviceToHost);
  printf("%f\n", *h_dummy);
}
