
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__global__ void saxpy(unsigned num_rd_streams, unsigned rd_stream_length, unsigned num_wr_streams, unsigned addr1, unsigned addr2, unsigned addr3, unsigned addr4, unsigned addr5, unsigned addr6, unsigned addr7, unsigned addr8, unsigned wr_stream_length)
{
	unsigned long long id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id <= 8) {
		for (unsigned i = 0; i < 1000 - 8; i += 8) {
			asm volatile ("st.shared.u32 [%%rd1], 4;");
			asm volatile ("st.shared.u32 [%%rd1], 4;");
			asm volatile ("st.shared.u32 [%%rd1], 4;");
			asm volatile ("st.shared.u32 [%%rd1], 4;");
			asm volatile ("st.shared.u32 [%%rd1], 4;");
			asm volatile ("st.shared.u32 [%%rd1], 4;");
			asm volatile ("st.shared.u32 [%%rd1], 4;");
			asm volatile ("st.shared.u32 [%%rd1], 4;");
		}
	}
}

int main(int argc, char *argv[])
{
	unsigned N = 1000;
	unsigned *d_x = (unsigned *)100;
	unsigned *h_x;
	h_x = (unsigned *)malloc(N*8*sizeof(unsigned));
	hipMemcpy(d_x, h_x, N*sizeof(unsigned), hipMemcpyHostToDevice);
	saxpy<<<1, 8>>>(0, 1000, 8, 100, 4100, 8100, 12100, 16100, 20100, 24100, 28100, 1000);
	hipMemcpy(h_x, d_x, N*8*sizeof(unsigned), hipMemcpyDeviceToHost);
	for (unsigned i = 0 ; i < 8000 ; i++) {
		printf("%u\n", *(h_x + i));
	}	
}
