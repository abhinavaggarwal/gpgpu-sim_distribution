
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__global__ void saxpy(unsigned num_rd_streams, unsigned addr1, unsigned addr2, unsigned addr3, unsigned addr4, unsigned addr5, unsigned addr6, unsigned addr7, unsigned addr8, unsigned rd_stream_length, unsigned num_wr_streams, unsigned wr_stream_length)
{
	int id = threadIdx.x;
	if (id <= 8) {
		for (int i = 0; i < 1000 - 8; i += 8) {
			unsigned a;
			asm ("ld.shared.u32 %0, [%1];" : "=r"(a) : "r"(id) );
			asm ("ld.shared.u32 %0, [%1];" : "=r"(a) : "r"(id) );
			asm ("ld.shared.u32 %0, [%1];" : "=r"(a) : "r"(id) );
			asm ("ld.shared.u32 %0, [%1];" : "=r"(a) : "r"(id) );
			asm ("ld.shared.u32 %0, [%1];" : "=r"(a) : "r"(id) );
			asm ("ld.shared.u32 %0, [%1];" : "=r"(a) : "r"(id) );
			asm ("ld.shared.u32 %0, [%1];" : "=r"(a) : "r"(id) );
			asm ("ld.shared.u32 %0, [%1];" : "=r"(a) : "r"(id) );
		}
	}
}

int main(int argc, char *argv[])
{
	int N = 1000;
	// Perform SAXPY on 1M elements
	unsigned *h_x = (unsigned *)malloc(N*sizeof(unsigned));
	unsigned *d_x = (unsigned *)100;
	unsigned *d_x_copy;
	hipMalloc(&d_x_copy, N*sizeof(unsigned));
	// cudaMalloc(&d_x, 2*sizeof(unsigned));
	for (int i = 1 ; i <= N ; i++)
		h_x[i-1] = (unsigned)i;
	hipMemcpy(d_x, h_x, N*sizeof(unsigned), hipMemcpyHostToDevice);
	saxpy<<<1, 8>>>(8, 100, 100, 100, 100, 100, 100, 100, 100, 1000, 0, 1000);
}
