
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__global__ void saxpy(unsigned num_rd_streams, unsigned addr1, unsigned addr2, unsigned addr3, unsigned addr4, unsigned addr5, unsigned addr6, unsigned addr7, unsigned addr8, unsigned rd_stream_length, unsigned num_wr_streams, unsigned wr_stream_length, unsigned *x)
{
	int id = threadIdx.x;
	if (id <= 0) {
		unsigned a;
		asm ("ld.shared.u32 %0, [%1];" : "=r"(a) : "r"(id) );
		x[0] = a;
		asm ("ld.shared.u32 %0, [%1];" : "=r"(a) : "r"(id) );
		x[1] = a;
	}
}

int main(int argc, char *argv[])
{
	int N = 1000;
	// Perform SAXPY on 1M elements
	unsigned *h_x = (unsigned *)malloc(N*sizeof(unsigned));
	unsigned *d_x = (unsigned *)100;
	unsigned *d_x_copy;
	hipMalloc(&d_x_copy, N*sizeof(unsigned));
	for (int i = 1 ; i <= N ; i++)
		h_x[i-1] = (unsigned)i;
	hipMemcpy(d_x, h_x, N*sizeof(unsigned), hipMemcpyHostToDevice);
	unsigned *h_dummy, *d_dummy;
	hipMalloc(&d_dummy, 2*sizeof(unsigned));
	h_dummy = (unsigned *)malloc(2*sizeof(unsigned));
	saxpy<<<1, 8>>>(8, 100, 100, 100, 100, 100, 100, 100, 100, 1000, 0, 1000, d_dummy);
	hipMemcpy(h_dummy, d_dummy, 2*sizeof(unsigned), hipMemcpyDeviceToHost);
	printf("%u\n", h_dummy[0]);
	printf("%u\n", h_dummy[1]);
}
